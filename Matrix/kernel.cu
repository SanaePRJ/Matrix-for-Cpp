﻿/* ---------------------------------------------------------------------------------------------
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 --------------------------------------------------------------------------------------------- */


#include <functional>
#include <iostream>
#include <random>
#include <time.h>
#include <chrono>
#include <thread>

#define _SANAE_MATRIX_ENABLE_CUDA_

#include "Matrix/Matrix"
#include "Test.hpp"


int main() {
    MulTestCPU();
    MulTestGPU();

    try {
        // コンストラクタによる初期化
        Sanae::Matrix<double> mat1{ {0.1, 0.2, 0.3}, {0.4, 0.5, 0.6}, {0.7, 0.8, 0.9} };
        std::cout << "行列 mat1:" << std::endl << mat1 << std::endl;

        Sanae::Matrix<double> mat2(std::make_pair(2, 3));
        std::cout << "行列 mat2 (2x3):" << std::endl << mat2 << std::endl;

        Sanae::Matrix<double> mat3(mat1);
        std::cout << "行列 mat3 (mat1のコピー):" << std::endl << mat3 << std::endl;

        // 演算子による操作
        mat1 += mat1;
        std::cout << "行列 mat1 加算後 (mat1 += mat1):" << std::endl << mat1 << std::endl;

        mat1 -= mat3;
        std::cout << "行列 mat1 減算後 (mat1 -= mat3):" << std::endl << mat1 << std::endl;

        mat1 *= 2;
        std::cout << "行列 mat1 スカラー倍後 (mat1 *= 2):" << std::endl << mat1 << std::endl;

        Sanae::Matrix<double> mat4 = mat1 * mat3;
        std::cout << "行列 mat4 (mat1 * mat3):" << std::endl << mat4 << std::endl;

        // 行列の転置
        Sanae::Matrix<double> mat5 = mat1.Transpose();
        std::cout << "行列 mat5 (mat1の転置):" << std::endl << mat5 << std::endl;

        // 行列のサイズ変更
        mat2.Resize(std::make_pair(3, 3));
        std::cout << "行列 mat2 サイズ変更後 (3x3):" << std::endl << mat2 << std::endl;

        // 単位行列とゼロ行列
        Sanae::Matrix<double> mat6 = Sanae::Matrix<double>::Identity(3);
        std::cout << "単位行列 mat6 (3x3):" << std::endl << mat6 << std::endl;

        Sanae::Matrix<double> mat7 = Sanae::Matrix<double>::Zero(3);
        std::cout << "ゼロ行列 mat7 (3x3):" << std::endl << mat7 << std::endl;

        // 行列式と逆行列
        double det = mat1.Det();
        std::cout << "行列 mat1 の行列式: " << det << std::endl;

        Sanae::Matrix<double> mat8 = mat1.Inverse();
        std::cout << "行列 mat8 (mat1の逆行列):" << std::endl << mat8 << std::endl;

        // キャスト
        Sanae::Matrix<float> matFloat = mat1;
        std::cout << "行列 matFloat (mat1からキャスト):" << std::endl << matFloat << std::endl;

        // アクセス
        std::cout << "要素 mat1[1][1]: " << mat1[1][1] << std::endl;

        // 譲渡
        Sanae::Matrix<double> mat9{ {10, 11, 12}, {13, 14, 15}, {16, 17, 18} };
        std::cout << "行列 mat9 (譲渡前):" << std::endl << mat9 << std::endl;

        mat9 << mat1;
        std::cout << "行列 mat9 (mat9 << mat1):" << std::endl << mat9 << std::endl;

        // 比較
        bool isEqual = (mat1 == mat3);
        std::cout << "比較 (mat1 == mat3): " << std::boolalpha << isEqual << std::endl;

        bool isNotEqual = (mat1 != mat3);
        std::cout << "比較 (mat1 != mat3): " << std::boolalpha << isNotEqual << "\n" << std::endl;

        // 1行取得
        std::cout << "1行分取得 mat9:";
        auto mat9Row = mat9.GetRowRef(0);

        for (auto& Cols : mat9Row)
            std::cout << std::setw(5) << Cols.get();
        std::cout << std::endl;

        // 1列取得
        std::cout << "1列分取得 mat9:";
        auto mat9Col = mat9.GetColRef(0);

        for (auto& Rows : mat9Col)
            std::cout << std::setw(5) << Rows.get();
        std::cout << "\n" << std::endl;

        //サイズ取得
        std::cout << "サイズ mat9:行" << mat9.Rows() << " 列:" << mat9.Cols() << std::endl;
    }
    catch (InvalidMatrix& e) {
        std::cerr << "行列エラー: " << e.what() << std::endl;
    }
    catch (std::exception& e) {
        std::cerr << "エラー: " << e.what() << std::endl;
    }

    return 0;
}